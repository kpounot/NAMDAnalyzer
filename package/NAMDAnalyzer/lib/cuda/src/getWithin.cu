#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define BLOCK_SIZE 384

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



__global__
void compWithin( float *allAtoms, int nbrAtoms, int nbrFrames, int *selAtoms, int sel_size,
                 int *out, float *cellDims, float distance, int frame )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    float squaredDist = distance * distance;

    for(int i=idx; i < sel_size; i+=BLOCK_SIZE)
        atomicExch( &out[ selAtoms[i]*nbrFrames + frame ], 1 );

    __syncthreads();

    float cD_x  = cellDims[3*frame];
    float cD_y  = cellDims[3*frame+1];
    float cD_z  = cellDims[3*frame+2];

    if(idx < sel_size) 
    {
        int selIdx = selAtoms[idx];

        float sel_x = allAtoms[3*nbrFrames*selIdx + 3*frame];
        float sel_y = allAtoms[3*nbrFrames*selIdx + 3*frame + 1];
        float sel_z = allAtoms[3*nbrFrames*selIdx + 3*frame + 2];

        for(int atom=0; atom < nbrAtoms; ++atom)
        {
            if(out[atom*nbrFrames + frame] == 0)
            {
                float atom_x = allAtoms[3*nbrFrames*atom + 3*frame];
                float dist_x = atom_x - sel_x;
                dist_x = dist_x - cD_x * roundf( dist_x / cD_x );
                if(dist_x > distance)
                    continue;

                float atom_y = allAtoms[3*nbrFrames*atom + 3*frame + 1];
                float dist_y = atom_y - sel_y;
                dist_y = dist_y - cD_y * roundf( dist_y / cD_y );
                if(dist_y > distance)
                    continue;

                float atom_z = allAtoms[3*nbrFrames*atom + 3*frame + 2];
                float dist_z = atom_z - sel_z;
                dist_z = dist_z - cD_z * roundf( dist_z / cD_z );

                float dist = dist_x*dist_x + dist_y*dist_y + dist_z*dist_z; 

                if(dist <= squaredDist)
                {
                    atomicExch( &out[atom*nbrFrames + frame], 1 );
                }
            }
        } // atom loop
    } // thread execution condition

}





void cu_getWithin_wrapper(  float *allAtoms, int nbrAtoms, int nbrFrames, int *selAtoms, int sel_size,
                            int *out, float *cellDims, float distance )
{
    // Copying atom1 matrix on GPU memory
    float *cu_allAtoms;
    size_t size = 3 * nbrAtoms * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_allAtoms, size) );
    gpuErrchk( hipMemcpy(cu_allAtoms, allAtoms, size, hipMemcpyHostToDevice) );

    // Copying selAtoms array on GPU memory
    int *cu_selAtoms;
    size = sel_size * sizeof(int);
    gpuErrchk( hipMalloc(&cu_selAtoms, size) );
    gpuErrchk( hipMemcpy(cu_selAtoms, selAtoms, size, hipMemcpyHostToDevice) );

    // Copying cellDims matrix on GPU memory
    float *cu_cellDims;
    size = 3 * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_cellDims, size) );
    gpuErrchk( hipMemcpy(cu_cellDims, cellDims, size, hipMemcpyHostToDevice) );

    // Copying out matrix on GPU memory
    int *cu_out;
    size = nbrAtoms * nbrFrames * sizeof(int);
    gpuErrchk( hipMalloc(&cu_out, size) );
    gpuErrchk( hipMemset(cu_out, 0, size) );


    int nbrBlocks = ceilf((float)sel_size / BLOCK_SIZE);

    for(int frame=0; frame < nbrFrames; ++frame)
    {
        compWithin<<<nbrBlocks, BLOCK_SIZE>>>(cu_allAtoms, nbrAtoms, nbrFrames, 
                                              cu_selAtoms, sel_size,
                                              cu_out, cu_cellDims, distance, frame);
        gpuErrchk( hipDeviceSynchronize() );
    }


    gpuErrchk( hipMemcpy(out, cu_out, size, hipMemcpyDeviceToHost) );


    hipFree(cu_allAtoms);
    hipFree(cu_selAtoms);
    hipFree(cu_cellDims);
    hipFree(cu_out);
}
