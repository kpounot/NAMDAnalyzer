#include <stdio.h>

#include <hip/hip_runtime.h>


#define BLOCK_SIZE 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void compIntScatFunc(float *atomPos, int atomPos_dim0, int atomPos_dim1, float *out,
                     float *qVecs, int qVecs_dim0, int qVecs_dim1, 
                     int nbrTS, int nbrTimeOri, int qValId) 
{
    int atomId  = blockDim.x * blockIdx.x + threadIdx.x;
    int TSIncr  = (atomPos_dim1 / nbrTS);

    extern __shared__ float s_qVecs[];

    if( atomId < atomPos_dim0 )
    {

        for(int i=0; i < 3*qVecs_dim0*qVecs_dim1; ++i)
            s_qVecs[i] = qVecs[i];

    }

    
    __syncthreads();


    if( atomId < atomPos_dim0 )
    {
        for(int dt=0; dt < nbrTS; ++dt)
        {
            int timeIncr = (float)(atomPos_dim1 - dt*TSIncr) / nbrTimeOri; 

            float sum_re = 0;
            float sum_im = 0;

            for(int t0=0; t0 < nbrTimeOri; ++t0)
            {
                // Gets indices
                int atom_tf_idx = 3 * (atomId*atomPos_dim1 + t0*timeIncr + dt*TSIncr); 
                int atom_t0_idx = 3 * (atomId*atomPos_dim1 + t0*timeIncr);

                // Computes distances for given timestep and atom
                float dist_0 = atomPos[atom_tf_idx] - atomPos[atom_t0_idx];
                float dist_1 = atomPos[atom_tf_idx+1] - atomPos[atom_t0_idx+1];
                float dist_2 = atomPos[atom_tf_idx+2] - atomPos[atom_t0_idx+2];

                for(int qVecId=0; qVecId < qVecs_dim1; ++qVecId)
                {

                    int qVec_idx = 3 * (qValId * qVecs_dim1 + qVecId);

                    float re = cos( s_qVecs[qVec_idx] * dist_0 
                                    + s_qVecs[qVec_idx+1] * dist_1
                                    + s_qVecs[qVec_idx+2] * dist_2 );

                    float im = sin( s_qVecs[qVec_idx] * dist_0 
                                    + s_qVecs[qVec_idx+1] * dist_1
                                    + s_qVecs[qVec_idx+2] * dist_2 );

                    sum_re += re;
                    sum_im += im;

                } // q vectors loop
            } // time origins loop 

            atomicAdd( &(out[2*(qValId*nbrTS + dt)]), sum_re / (nbrTS*qVecs_dim1) );
            atomicAdd( &(out[2*(qValId*nbrTS + dt) + 1]), sum_im / (nbrTS*qVecs_dim1) );

        } // time increments loop

    } // condition on atom index

}





void cu_compIntScatFunc_wrapper(float *atomPos, int atomPos_dim0, int atomPos_dim1, int atomPos_dim2, 
                                float *qVecs, int qVecs_dim0, int qVecs_dim1, int qVecs_dim2, 
                                float *out, int nbrTS, int nbrTimeOri)
{
    // Copying atomPos matrix on GPU memory
    float *cu_atomPos;
    size_t size_atomPos = atomPos_dim0 * atomPos_dim1 * atomPos_dim2 * sizeof(float);
    gpuErrchk( hipMalloc(&cu_atomPos, size_atomPos) );
    gpuErrchk( hipMemcpy(cu_atomPos, atomPos, size_atomPos, hipMemcpyHostToDevice) );

    // Copying qVecs matrix on GPU memory
    float *cu_qVecs;
    size_t size_qVecs = qVecs_dim0 * qVecs_dim1 * qVecs_dim2 * sizeof(float);
    gpuErrchk( hipMalloc(&cu_qVecs, size_qVecs) );
    gpuErrchk( hipMemcpy(cu_qVecs, qVecs, size_qVecs, hipMemcpyHostToDevice) );

    // Copying out matrix on GPU memory
    float *cu_out;
    size_t size_out = 2 * qVecs_dim0 * nbrTS * sizeof(float);
    gpuErrchk( hipMalloc(&cu_out, size_out) );
    gpuErrchk( hipMemset(cu_out, 0, size_out) );

    int nbrBlocks = ceil((float)atomPos_dim0 / BLOCK_SIZE);
    int sharedMemSize = sizeof(float) * 3 * qVecs_dim0 * qVecs_dim1; 


    // Starts computation of intermediate scattering function
    for(int qValId=0; qValId < qVecs_dim0; ++qValId)
    {
        compIntScatFunc<<<nbrBlocks, BLOCK_SIZE, sharedMemSize>>>(cu_atomPos, atomPos_dim0, 
                                                            atomPos_dim1, cu_out, cu_qVecs, 
                                                            qVecs_dim0, qVecs_dim1, nbrTS, 
                                                            nbrTimeOri, qValId);
        gpuErrchk( hipDeviceSynchronize() );
    }



    hipMemcpy(out, cu_out, size_out, hipMemcpyDeviceToHost);

    for(int i=0; i < 2*nbrTS*qVecs_dim0; ++i)
    {
        out[i] /= atomPos_dim0;
    }

    hipFree(cu_atomPos);
    hipFree(cu_qVecs);
    hipFree(cu_out);
}
