#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>


#define BLOCK_SIZE 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*__________________________________________________

  Hydrogen bond autocorrelation
  __________________________________________________ */

__global__
void d_getHBCorr(float *acceptors, int size_acceptors, int nbrFrames,
                        float *donors, int size_donors, float *hydrogens, int size_hydrogens, 
                        float *cellDims,
                        float *out, int maxTime, int step, int nbrTimeOri,
                        float maxR, float cosAngle, int continuous )
{
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;


    for(int dt=0; dt < nbrFrames; ++dt)
    {
        if( row < size_acceptors && col < size_donors )
        {
            float cD_x = cellDims[3*dt];
            float cD_y = cellDims[3*dt + 1];
            float cD_z = cellDims[3*dt + 2];

            // Computes distances for given timestep and atom
            float hyd_x = hydrogens[3 * nbrFrames * col + 3 * dt];
            float acc_x = acceptors[3 * nbrFrames * row + 3 * dt]; 
            float h_acc_x = hyd_x - acc_x;
            h_acc_x = h_acc_x - cD_x * roundf(h_acc_x / cD_x);

            float hyd_y = hydrogens[3 * nbrFrames * col + 3 * dt + 1]; 
            float acc_y = acceptors[3 * nbrFrames * row + 3 * dt + 1]; 
            float h_acc_y = hyd_y - acc_y;
            h_acc_y = h_acc_y - cD_y * roundf(h_acc_y / cD_y);

            float hyd_z = hydrogens[3 * nbrFrames * col + 3 * dt + 2]; 
            float acc_z = acceptors[3 * nbrFrames * row + 3 * dt + 2]; 
            float h_acc_z = hyd_z - acc_z;
            h_acc_z = h_acc_z - cD_z * roundf(h_acc_z / cD_z);


            float don_x = donors[3 * nbrFrames * col + 3 * dt]; 
            float don_y = donors[3 * nbrFrames * col + 3 * dt + 1]; 
            float don_z = donors[3 * nbrFrames * col + 3 * dt + 2]; 

            float acc_d_x = acc_x - don_x;
            float acc_d_y = acc_y - don_y;
            float acc_d_z = acc_z - don_z;


            float dist = sqrtf(h_acc_x*h_acc_x + h_acc_y*h_acc_y + h_acc_z*h_acc_z); 

            float angle = (h_acc_x * acc_d_x + h_acc_y * acc_d_y + h_acc_z * acc_d_z);
            angle /= ( dist * sqrtf(acc_d_x*acc_d_x + acc_d_y*acc_d_y + acc_d_z*acc_d_z) );

            if(dist <= maxR && angle <= cosAngle)
                atomicAdd( &out[dt], 1 );

            else
            {
                if(dt == 0) 
                    break;

                if(continuous==1)
                    break;
            }

        } // if loop, matrix boundaries
    } // time steps loop

}





void getHBCorr_wrapper(  float *acceptors, int size_acceptors, int nbrFrames,
                                float *donors, int size_donors,
                                float *hydrogens, int size_hydrogens, 
                                float *cellDims,
                                float *out, int maxTime, int step, 
                                int nbrTimeOri, float maxR, float minAngle, int continuous )
{

    float cosAngle = cosf(minAngle);

    // Copying acceptors matrix on GPU memory
    float *cu_acceptors;
    size_t size = 3 * size_acceptors * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_acceptors, size) );
    gpuErrchk( hipMemcpy(cu_acceptors, acceptors, size, hipMemcpyHostToDevice) );

    // Copying donors matrix on GPU memory
    float *cu_donors;
    size = 3 * size_donors * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_donors, size) );
    gpuErrchk( hipMemcpy(cu_donors, donors, size, hipMemcpyHostToDevice) );

    // Copying hydrogens matrix on GPU memory
    float *cu_hydrogens;
    size = 3 * size_hydrogens * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_hydrogens, size) );
    gpuErrchk( hipMemcpy(cu_hydrogens, hydrogens, size, hipMemcpyHostToDevice) );

    // Copying hydrogens matrix on GPU memory
    float *cu_cellDims;
    size = 3 * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_cellDims, size) );
    gpuErrchk( hipMemcpy(cu_cellDims, cellDims, size, hipMemcpyHostToDevice) );

    // Copying out matrix on GPU memory
    float *cu_out;
    size = nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_out, size) );
    gpuErrchk( hipMemset(cu_out, 0, size) );



    dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE, 1 );
    dim3 dimGrid( ceil( (float)size_donors/BLOCK_SIZE), ceil( (float)size_acceptors/BLOCK_SIZE), 1);

    d_getHBCorr<<<dimGrid, dimBlock>>>(cu_acceptors, size_acceptors, 
                                       nbrFrames, cu_donors, 
                                       size_donors, cu_hydrogens, size_hydrogens, 
                                       cu_cellDims,
                                       cu_out, maxTime, step, nbrTimeOri, 
                                       maxR, cosAngle, continuous);
    gpuErrchk( hipDeviceSynchronize() );

    // Copying result back into host memory
    gpuErrchk( hipMemcpy(out, cu_out, size, hipMemcpyDeviceToHost) );

    hipFree(cu_acceptors);
    hipFree(cu_donors);
    hipFree(cu_hydrogens);
    hipFree(cu_cellDims);
    hipFree(cu_out);
}





/*__________________________________________________

  Hydrogen bond number
  __________________________________________________ */


__global__
void d_getHBNbr(float *acceptors, int size_acceptors, int nbrFrames,
                        float *donors, int size_donors, float *hydrogens, int size_hydrogens, 
                        float *cellDims, float *out, float maxR, float cosAngle,
                        int frame )
{
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;


    if( row < size_acceptors && col < size_donors )
    {
        float cD_x = cellDims[3*frame];
        float cD_y = cellDims[3*frame + 1];
        float cD_z = cellDims[3*frame + 2];

        // Computes distances for given timestep and atom
        float hyd_x = hydrogens[3 * nbrFrames * col + 3 * frame];
        float acc_x = acceptors[3 * nbrFrames * row + 3 * frame]; 
        float h_acc_x = hyd_x - acc_x;
        h_acc_x = h_acc_x - cD_x * roundf(h_acc_x / cD_x);

        float hyd_y = hydrogens[3 * nbrFrames * col + 3 * frame + 1]; 
        float acc_y = acceptors[3 * nbrFrames * row + 3 * frame + 1]; 
        float h_acc_y = hyd_y - acc_y;
        h_acc_y = h_acc_y - cD_y * roundf(h_acc_y / cD_y);

        float hyd_z = hydrogens[3 * nbrFrames * col + 3 * frame + 2]; 
        float acc_z = acceptors[3 * nbrFrames * row + 3 * frame + 2]; 
        float h_acc_z = hyd_z - acc_z;
        h_acc_z = h_acc_z - cD_z * roundf(h_acc_z / cD_z);


        float don_x = donors[3 * nbrFrames * col + 3 * frame]; 
        float don_y = donors[3 * nbrFrames * col + 3 * frame + 1]; 
        float don_z = donors[3 * nbrFrames * col + 3 * frame + 2]; 

        float acc_d_x = acc_x - don_x;
        float acc_d_y = acc_y - don_y;
        float acc_d_z = acc_z - don_z;


        float dist = sqrtf(h_acc_x*h_acc_x + h_acc_y*h_acc_y + h_acc_z*h_acc_z); 

        float angle = (h_acc_x * acc_d_x + h_acc_y * acc_d_y + h_acc_z * acc_d_z);
        angle /= ( dist * sqrtf(acc_d_x*acc_d_x + acc_d_y*acc_d_y + acc_d_z*acc_d_z) );

        if(dist <= maxR && angle <= cosAngle)
            atomicAdd( &out[frame], 1 );


    } // if loop, matrix boundaries

}



void getHBNbr_wrapper(  float *acceptors, int size_acceptors, int nbrFrames,
                                float *donors, int size_donors,
                                float *hydrogens, int size_hydrogens, 
                                float *cellDims,
                                float *out, 
                                float maxR, float minAngle )
{

    float cosAngle = cosf(minAngle);

    // Copying acceptors matrix on GPU memory
    float *cu_acceptors;
    size_t size = 3 * size_acceptors * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_acceptors, size) );
    gpuErrchk( hipMemcpy(cu_acceptors, acceptors, size, hipMemcpyHostToDevice) );

    // Copying donors matrix on GPU memory
    float *cu_donors;
    size = 3 * size_donors * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_donors, size) );
    gpuErrchk( hipMemcpy(cu_donors, donors, size, hipMemcpyHostToDevice) );

    // Copying hydrogens matrix on GPU memory
    float *cu_hydrogens;
    size = 3 * size_hydrogens * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_hydrogens, size) );
    gpuErrchk( hipMemcpy(cu_hydrogens, hydrogens, size, hipMemcpyHostToDevice) );

    // Copying hydrogens matrix on GPU memory
    float *cu_cellDims;
    size = 3 * nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_cellDims, size) );
    gpuErrchk( hipMemcpy(cu_cellDims, cellDims, size, hipMemcpyHostToDevice) );

    // Copying out matrix on GPU memory
    float *cu_out;
    size = nbrFrames * sizeof(float);
    gpuErrchk( hipMalloc(&cu_out, size) );
    gpuErrchk( hipMemset(cu_out, 0, size) );


    dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE, 1 );
    dim3 dimGrid( ceil( (float)size_donors/BLOCK_SIZE), ceil( (float)size_acceptors/BLOCK_SIZE), 1);

    for(int frame=0; frame < nbrFrames; ++frame)
    {
        printf("Processing frame %i of %i...        \r", frame+1, nbrFrames);

        d_getHBNbr<<<dimGrid, dimBlock>>>(cu_acceptors, size_acceptors, 
                                           nbrFrames, cu_donors, 
                                           size_donors, cu_hydrogens, size_hydrogens, 
                                           cu_cellDims, cu_out, maxR, cosAngle, frame);
        gpuErrchk( hipDeviceSynchronize() );
    }

    // Copying result back into host memory
    gpuErrchk( hipMemcpy(out, cu_out, size, hipMemcpyDeviceToHost) );

    hipFree(cu_acceptors);
    hipFree(cu_donors);
    hipFree(cu_hydrogens);
    hipFree(cu_cellDims);
    hipFree(cu_out);
}
